#include "hip/hip_runtime.h"
#include "fluid_sim.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// CUDA kernels for fluid dynamics

__global__ void advect(float* field, float* field0, int gridSize, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= gridSize || j >= gridSize) return;

    // Backtracking particles for advection
    float x = i - field0[i + j * gridSize] * dt;
    float y = j - field0[i + j * gridSize] * dt;

    x = max(0.0f, min((float)gridSize - 1, x));
    y = max(0.0f, min((float)gridSize - 1, y));

    int i0 = (int)x;
    int i1 = i0 + 1;
    int j0 = (int)y;
    int j1 = j0 + 1;

    float s1 = x - i0;
    float s0 = 1 - s1;
    float t1 = y - j0;
    float t0 = 1 - t1;

    field[i + j * gridSize] = s0 * (t0 * field0[i0 + j0 * gridSize] + t1 * field0[i0 + j1 * gridSize]) +
                              s1 * (t0 * field0[i1 + j0 * gridSize] + t1 * field0[i1 + j1 * gridSize]);
}

__global__ void diffuse(float* field, float* field0, float diffusion, int gridSize, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= gridSize || j >= gridSize) return;

    for (int k = 0; k < 20; k++) { // Gauss-Seidel relaxation
        field[i + j * gridSize] =
            (field0[i + j * gridSize] + diffusion * dt *
             (field[i - 1 + j * gridSize] + field[i + 1 + j * gridSize] +
              field[i + (j - 1) * gridSize] + field[i + (j + 1) * gridSize])) /
            (1 + 4 * diffusion * dt);
    }
}

// Host function to run the simulation
void simulate_fluid(const FluidSimParams& params, int steps) {
    int gridSize = params.gridSize;
    int gridSize2 = gridSize * gridSize;
    size_t gridBytes = gridSize2 * sizeof(float);

    float *field, *field0;
    hipMalloc(&field, gridBytes);
    hipMalloc(&field0, gridBytes);

    hipMemset(field, 0, gridBytes);
    hipMemset(field0, 0, gridBytes);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((gridSize + 15) / 16, (gridSize + 15) / 16);

    for (int step = 0; step < steps; step++) {
        // Advection step
        advect<<<numBlocks, threadsPerBlock>>>(field, field0, gridSize, params.dt);

        // Diffusion step
        diffuse<<<numBlocks, threadsPerBlock>>>(field, field0, params.diffusion, gridSize, params.dt);

        hipMemcpy(field0, field, gridBytes, hipMemcpyDeviceToDevice);
    }

    hipFree(field);
    hipFree(field0);

    write_grid_to_csv(field, gridSize, "fluid_sim_output.csv");
}

void write_grid_to_csv(const float* field, int gridSize, const std::string& filename) {
    std::vector<float> hostData(gridSize * gridSize);
    hipMemcpy(hostData.data(), field, gridSize * gridSize * sizeof(float), hipMemcpyDeviceToHost);

    std::ofstream file(filename);
    for (int j = 0; j < gridSize; j++) {
        for (int i = 0; i < gridSize; i++) {
            file << hostData[i + j * gridSize];
            if (i < gridSize - 1) file << ",";
        }
        file << "\n";
    }
    file.close();
}