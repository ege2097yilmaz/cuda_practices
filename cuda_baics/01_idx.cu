
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c){
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(){
    const int SIZE = 10;
    int a[SIZE], b[SIZE], c[SIZE];
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, SIZE * sizeof(int));
    hipMalloc((void**)&d_b, SIZE * sizeof(int));
    hipMalloc((void**)&d_c, SIZE * sizeof(int));

    // initialize the 
    for(int i = 0; i < SIZE; i++){
        a[i] = i;
        b[i] = i * i;
    }

    hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, SIZE>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++){
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl; 
    }
     
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}